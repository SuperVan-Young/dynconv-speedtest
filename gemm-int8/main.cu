
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>
#include <hipblas.h>

using namespace std;

#define CUDA_CALL(call)                                                  \
do {                                                                     \
    const hipError_t error_code = call;                                 \
    if (error_code != hipSuccess) {                                     \
        printf("CUDA Error:\n");                                         \
        printf("    File:       %s\n", __FILE__);                        \
        printf("    Line:       %d\n", __LINE__);                        \
        printf("    Error Code: %d\n", error_code);                      \
        printf("    Error Text: %s\n", hipGetErrorString(error_code));  \
        exit(1);                                                         \
    }                                                                    \
} while (0)

#define KERNEL_CUBLASGEMMEX 1

typedef struct GemmArgs {
    int M, N, K;
    void *alpha, *beta, *A, *B, *C;
} GemmArgs_t;

void createSample(int M, int N, int K, GemmArgs_t **args) {
    *args = (GemmArgs_t*)malloc(sizeof(GemmArgs_t));
    (*args)->M = M;
    (*args)->N = N;
    (*args)->K = K;
    
    // ignored memory initilization

    // (*args)->alpha = malloc(sizeof(int32_t));
    // (*args)->beta  = malloc(sizeof(int32_t));
    // (*args)->A     = malloc(sizeof(int32_t)*M*K);
    // (*args)->B     = malloc(sizeof(int32_t)*K*N);
    // (*args)->C     = malloc(sizeof(int32_t)*M*N);
}

void deleteSample(GemmArgs_t *args) {
    // ignored memory initialization 
    free(args);
}

float speedTest(int kernel, GemmArgs_t *sample) {
    // prepare device data
    int M = sample->M;
    int N = sample->N;
    int K = sample->K;
    int32_t h_alpha = 1, h_beta = 1;
    void *d_alpha, *d_beta, *d_A, *d_B, *d_C;
    CUDA_CALL(hipMalloc(&d_alpha, sizeof(int32_t)));
    CUDA_CALL(hipMalloc(&d_beta, sizeof(int32_t)));
    CUDA_CALL(hipMalloc(&d_A, sizeof(int32_t)*M*K));
    CUDA_CALL(hipMalloc(&d_B, sizeof(int32_t)*K*N));
    CUDA_CALL(hipMalloc(&d_C, sizeof(int32_t)*M*N));
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // ignored memory copy
    const int one = 1;
    hipMemcpy(d_alpha, &one, sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_beta , &one, sizeof(int32_t), hipMemcpyHostToDevice);

    // start timing
    CUDA_CALL(hipDeviceSynchronize());
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    CUDA_CALL(hipEventRecord(start));
    hipEventQuery(start);
    CUDA_CALL(hipDeviceSynchronize());

    //=======================  Timing Code Block  ==============================

    switch(kernel) {
        case KERNEL_CUBLASGEMMEX:
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &h_alpha, d_A, 
                         HIP_R_8I, M, d_B, HIP_R_8I, K, &h_beta, d_C, HIP_R_32I,
                         M, HIP_R_32I, HIPBLAS_GEMM_DEFAULT);
            break;
        default:
            break;
    }
    
    //==========================================================================

    // end timing
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    float elapsed_time;
    CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, stop));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));

    // clean up the memory
    CUDA_CALL(hipFree(d_alpha));
    CUDA_CALL(hipFree(d_beta));
    CUDA_CALL(hipFree(d_A));
    CUDA_CALL(hipFree(d_B));
    CUDA_CALL(hipFree(d_C));

    return elapsed_time;
}

int main(int argc, char *argv[]) {
    int kernel = 0;
    int opt;
    const char *optstring = "k:";

    while ((opt = getopt(argc, argv, optstring)) != -1) {
        switch (opt) {
            case 'k':
                sscanf(optarg, "%d", &kernel);
                break;
            default:
                printf("Unknown argument %c\n", opt);
                exit(1);
        }
    }

    GemmArgs_t *args;
    for (int i = 1; i <= 30; i++) {
        int len = i << 10;
        createSample(len, len, len, &args);

        float elapsed_time = speedTest(kernel, args);
        float gflops = 2.*1e-6 * len * len * len / elapsed_time;
        printf("(%5d): %.5f ms    %f GFLOPS\n", len, elapsed_time, gflops);

        deleteSample(args);
    }
}